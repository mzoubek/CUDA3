#include "hip/hip_runtime.h"
#include "cuda_img.h"
#include "rotate.h"
#include <hip/hip_runtime.h>

__global__
void kernel_rotate( const CudaImg src, CudaImg dst, KernelRot kr )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= dst.m_size.x || y >= dst.m_size.y ) return;

	// Souradnice vuci stredu
	int cx = x - dst.m_size.x / 2;
	int cy = y - dst.m_size.y / 2;

	// Rotace
	float ox = kr.c * cx - kr.s * cy;
	float oy = kr.s * cx + kr.c * cy;

	// zpet do src-coords
	int sx = int( ox + src.m_size.x / 2 );
	int sy = int( oy + src.m_size.y / 2 );
	if ( sx < 0 || sx >= src.m_size.x || sy < 0 || sy >= src.m_size.y ) return;

	int pixelDst = y * dst.m_size.x + x;
	int pixelSrc = sy * src.m_size.x + x;
	dst.m_p_uchar4[ pixelDst ] = src.m_p_uchar4[ pixelSrc ];
}

void cu_run_rotate( const CudaImg &src, CudaImg &dst, const KernelRot &kr )
{
	hipError_t cerr;

	dim3 threads( 32, 32 );
	dim3 blocks( ( dst.m_size.x + threads.x - 1 ) / threads.x,
			( dst.m_size.y + threads.y - 1 ) / threads.y );
	kernel_rotate<<<blocks, threads>>>( src, dst, kr );
	
	if ( ( cerr = hipGetLastError() ) != hipSuccess )
	{
		printf( "CUDA error[%d]: %s\n", __LINE__, hipGetErrorString( cerr ) );
	}

	hipDeviceSynchronize();
}
